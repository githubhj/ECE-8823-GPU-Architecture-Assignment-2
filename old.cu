#include "hip/hip_runtime.h"
//============================================================================
// Name        : convolution2d.cu
// Author      : Harshit Jain
// Class	   : ECE 8823
// GTID		   : 903024992
// Assignment  : Assignment 1
// Copyright   : Public
// Description : 2D Convolution in CUDA
//============================================================================

#include <iostream>
#include <fstream>
#include <cassert>
#include <sstream>
#include "pgma_io.hpp"
#include <vector>
#include <string>
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define TILE_WIDTH 32
#define KERNEL_SIZE 7

//extern __shared__ int s[];

#define checkCudaError(status) { \
	if(status != hipSuccess) { \
		std::cout << "CUDA Error " << __FILE__ << ", " << __LINE__ \
			<< ": " << hipGetErrorString(status) << "\n"; \
		exit(-1); \
	} \
}

__constant__ int gpuKernel[31*31];


__global__ void convolutionGPU(int* inputImage, int* outputImage, int imageWidth, int kernelSize, int totalVal) {

	//ADD CODE HERE
	//Shared memory of size TILE_WIDTH plus apron width on top and bottom
	extern __shared__ int sharedImageData[];
	
	//get kernel radius
	int kRadius = kernelSize/2;
	
	//get particular thread data location in input image
	int threadDataLoc = threadIdx.x + blockIdx.x*blockDim.x + threadIdx.y*imageWidth + (blockIdx.y*blockDim.y)*imageWidth;
	
	//get thread x,y coordinates
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	
	
	//Let each thread write to four locations in shared memory
	int xUleft = x - kRadius;
	int yUleft = y - kRadius;
	
	int xLleft = x - kRadius;
	int yLleft = y + kRadius;
	
	int xURight = x + kRadius;
	int yURight = y - kRadius;
	
	int xLRight = x + kRadius;
	int yLRight = y + kRadius;
	
	
	if(xUleft >= 0 && yUleft >= 0){
		sharedImageData[threadIdx.x + ((TILE_WIDTH + kernelSize-1)*threadIdx.y)] = inputImage[threadDataLoc - kRadius - (imageWidth*kRadius)];
		if(blockIdx.x == 0 && blockIdx.y==0 && threadIdx.x ==1 && threadIdx.y ==1){
			printf("%d",sharedImageData[threadIdx.x + ((TILE_WIDTH + kernelSize-1)*threadIdx.y)] );
		}
	}
	else{
		sharedImageData[threadIdx.x + ((TILE_WIDTH + kernelSize-1)*threadIdx.y)] = 0;
	}
		
	
	if(xLleft >= 0 && (yLleft <= (imageWidth -1))){
		sharedImageData[threadIdx.x + ((TILE_WIDTH+kernelSize-1)*(threadIdx.y +kRadius))] = inputImage[threadDataLoc - kRadius + (imageWidth*kRadius)];
	}
	else{
		sharedImageData[threadIdx.x + ((TILE_WIDTH+kernelSize-1)*(threadIdx.y + kRadius))] = 0;
	}	
	
	if(xURight <= (imageWidth -1) && yURight >= 0){
		sharedImageData[threadIdx.x + kRadius+ ((TILE_WIDTH+kernelSize-1)*threadIdx.y)] = inputImage[threadDataLoc + kRadius - (imageWidth*kRadius)];
	}
	else{
		sharedImageData[threadIdx.x + kRadius+ ((TILE_WIDTH+kernelSize-1)*threadIdx.y)] = 0;
	}
	
	
	if((xLRight  <= (imageWidth -1)) && (yLRight <= (imageWidth -1))){
		sharedImageData[threadIdx.x + kRadius + ((TILE_WIDTH+kernelSize-1)*(threadIdx.y + kRadius))] = inputImage[threadDataLoc + kRadius + (imageWidth*kRadius)];
	}
	else{
		sharedImageData[threadIdx.x + kRadius + ((TILE_WIDTH+kernelSize-1)*(threadIdx.y + kRadius))] = 0;
	}
	
	if(blockIdx.x == 0 && blockIdx.y==0 && threadIdx.x ==1 && threadIdx.y ==1){
		printf("%d,%d,%d,%d\n",sharedImageData[threadIdx.x + ((TILE_WIDTH + kernelSize-1)*threadIdx.y)], sharedImageData[threadIdx.x + ((TILE_WIDTH+kernelSize-1)*(threadIdx.y +kRadius))], sharedImageData[threadIdx.x + kRadius+ ((TILE_WIDTH+kernelSize-1)*threadIdx.y)], sharedImageData[threadIdx.x + kRadius + ((TILE_WIDTH+kernelSize-1)*(threadIdx.y + kRadius))]);
		printf("%d\n",threadDataLoc);
	}
	
	__syncthreads();

	/*
	int value = 0;
	for (int kRow = -kRadius; kRow <= kRadius; kRow++)
		for (int kCol = -kRadius; kCol <= kRadius; kCol++)
			value += sharedImageData[(threadIdx.x+kRadius) + kCol+ (threadIdx.y + kRadius + kRow)*blockDim.x] * gpuKernel[(kRadius + kRow)*kernelSize + kRadius + kCol];
	outputImage[threadDataLoc] = value/totalVal;
	*/
	
	outputImage[threadDataLoc] = sharedImageData[threadIdx.x + kRadius + (threadIdx.y+kRadius)*(TILE_WIDTH+kernelSize-1)];
}


class PGM
{
public:
	PGM() : N(0), ptr(NULL) {}
	PGM(const PGM &rhs) : N(0), ptr(NULL)
	{
		copy(rhs);
	}
	~PGM() {
		if (ptr != NULL) {
			delete [] ptr;
		}
	}
	PGM& operator=(const PGM &rhs)
	{
		if (this == &rhs)
			return *this;
		return copy(rhs);
	}
	PGM& copy(const PGM &rhs)
	{
		if (ptr != NULL)
		{
			delete [] ptr;
		}
		N = rhs.N;
		size_t imageSize = N * N * sizeof *(rhs.ptr);
		ptr = new int[imageSize];
		memcpy(ptr, rhs.ptr, imageSize);
		return *this;
	}
	bool operator==(const PGM &rhs) const
	{
		if (N == rhs.N) {
			for(int i = 0; i < N * N; i++)
			{
				if (ptr[i] != rhs.ptr[i])
				{
					return false;
				}
			}
		} else {
			return false;
		}
		return true;
	}
	int N;
	int *ptr;
};

PGM getImage(std::string fileName)
{
	PGM image;
	int x, y, maxVal;
	pgma_read(fileName, x, y, maxVal, &(image.ptr));
	assert(x == y);
	image.N = x;
	return image;
}

int main(int argc, char** argv)
{
	assert(argc > 2);
	std::vector<std::string> args;
	std::copy(argv+1, argv + argc, std::back_inserter(args));
	
	std::string fileName = args[1];
	PGM image = getImage(fileName);
	PGM hostOutput = image;
	
	//construct kxk filter
	std::cout << "Constructing kernel:\n";
	int k = atoi(args[0].c_str());
	assert(k % 2 == 1);
	int *kernel = new int[k*k*sizeof(int)];
	int totalVal = 0;
	for (int row = 0; row < k; row++)
	{
		for (int col = 0; col < k; col++)
		{
			int colVal = (col < (k/2+1)) ? col+1 : k-col;
			int rowVal = (row < (k/2+1)) ? row+1 : k-row;
			kernel[row * k + col] = colVal + rowVal;
			totalVal += colVal + rowVal;
			std::cout << kernel[row*k + col] << " ";
		}
		std::cout << "\n";
	}

	PGM gpuOutput = image;
	std::cout << "Image Width : " << image.N << std::endl;
/////////////////////////////////////////////////////////////////	 
// Insert CUDA launch code here
/////////////////////////////////////////////////////////////////
	int device;
	int * gpuInputImage, * gpuOutputImage;
	
	checkCudaError(hipSetDevice(5));

	checkCudaError(hipGetDevice(&device));
	hipDeviceProp_t prop;
	checkCudaError(hipGetDeviceProperties(&prop, device));
	std::cout << "Device " << device << ": " << prop.name << "\n";
	std::cout << "GPU/SM Cores: " << prop.multiProcessorCount << "\n";
	std::cout << "Compute Capability: " << prop.major << "." << prop.minor << "\n";
	std::cout << "Shared Memory per Block: " << (prop.sharedMemPerBlock>>10) << "\n";
	
	
	checkCudaError(hipMalloc(&gpuInputImage, image.N * image.N * sizeof(int)));
	std::cout << "Woks" << std::endl;
    checkCudaError(hipMalloc(&gpuOutputImage, image.N * image.N * sizeof(int)));
    
    checkCudaError(hipMemcpy(gpuInputImage, image.ptr, image.N * image.N * sizeof(int), hipMemcpyHostToDevice));
	checkCudaError(hipMemcpy(gpuOutputImage, gpuOutput.ptr, image.N * image.N * sizeof(int), hipMemcpyHostToDevice));
	checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gpuKernel), kernel, k * k * sizeof(int)));
    
    const int GRID_SIZE = ceil((float)image.N/TILE_WIDTH);
    std::cout << GRID_SIZE << std::endl;
	const int CTA_SIZE = TILE_WIDTH;
    
    std::cout << "Image size: " << image.N << "X" << image.N << std::endl << "Threads per block: " << CTA_SIZE << "X" << CTA_SIZE << std::endl << "Blocks: " << GRID_SIZE << "X" << GRID_SIZE << std::endl;
    
    dim3 dimBlock(CTA_SIZE,CTA_SIZE,1);
    dim3 dimGrid(GRID_SIZE,GRID_SIZE);
	
	std::cout << "Running GPU kernel\n\n";
	//use k, kernel, image.N and image.ptr as your inputs
	//copy output to gpuOutput.ptr, data is already allocated
	//make sure the dimensions of the image are the same
	
	int shared_memory = sizeof(int)*(TILE_WIDTH+k-1)*(TILE_WIDTH+k-1);
	printf("Shared Memory: %d\n",shared_memory>>10);
	
	hipProfilerStart();
	convolutionGPU<<<dimGrid, dimBlock, shared_memory>>>(gpuInputImage, gpuOutputImage, image.N, k, totalVal);
	hipProfilerStop();
	
	checkCudaError(hipDeviceSynchronize());
	hipMemcpy(gpuOutput.ptr, gpuOutputImage, image.N * image.N * sizeof(int), hipMemcpyDeviceToHost);
	printf("%d\n",image.ptr[0]);
	
/////////////////////////////////////////////////////////////////

	//CPU convolution
	std::cout << "Running host kernel\n\n";
	int kRadius = k/2;
	for (int row = 0; row < image.N; row++)
	{
		for (int col = 0; col < image.N; col++)
		{
			//sample from neighbor pixels 
			int index = row * image.N + col;
			int value = 0;
			for (int kRow = -kRadius; kRow <= kRadius; kRow++)
			{
				//image bounds check
				if (row+kRow < 0 || row+kRow >= image.N)
					continue;
				for (int kCol = -kRadius; kCol <= kRadius; kCol++)
				{
					//image bounds check
					if (col+kCol < 0 || col+kCol >= image.N)
						continue;
					value += kernel[(kRadius + kRow)*k + kRadius + kCol] * image.ptr[index + kRow*image.N + kCol];
				}
			}
			hostOutput.ptr[index] = value / totalVal;
		}
	}

	std::cout << "Comparing results:\n";
	bool passed = hostOutput == gpuOutput;
	std::string resultString = (passed) ? "Passed\n" : "Failed\n";
	std::cout << resultString;

	std::cout << "Writing image outputs: output_host.pgm output_gpu.pgm\n";
	std::string outputFileName = "output_host.pgm";
	pgma_write(outputFileName, image.N, image.N, hostOutput.ptr);
	outputFileName = "output_gpu.pgm";
	pgma_write(outputFileName, image.N, image.N, gpuOutput.ptr);

	delete(kernel);
}
